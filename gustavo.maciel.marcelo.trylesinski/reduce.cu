#include "hip/hip_runtime.h"
#include "reduce.cuh"
#include "mtrutils.h"

#include <cmath>

#define R(i,j) R[(i) * matrices_dim + (j)]
#define A(i,j) A[(i) * matrices_dim + (j)]
#define B(i,j) B[(i) * matrices_dim + (j)]

// 
static void binary_reduce(matrix A, matrix B, matrix R, int matrices_dim)
{
    for (int i = 0; i < matrices_dim; i++)
        for (int j = 0; j < matrices_dim; j++)
            R(i,j) = fmin(A(i,j), B(i,j));
}

// 
__global__ void cuda_reduce_kernel(matrix *matrices, int matrices_num,
                                   matrix result, int matrices_dim)
{
}

void sequential_reduce(matrix *matrices, int matrices_num, matrix result,
                       int matrices_dim)
{
    binary_reduce(matrices[0], matrices[1], result, matrices_dim);

    for (int i = 2; i < matrices_num; i++)
        binary_reduce(result, matrices[i], result, matrices_dim);
}

void cuda_reduce(matrix *matrices, int matrices_num, matrix result,
                 int matrices_dim)
{
}
