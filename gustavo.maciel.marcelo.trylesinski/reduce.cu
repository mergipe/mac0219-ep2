#include "hip/hip_runtime.h"
#include "reduce.cuh"
#include "mtrutils.h"

#include <cmath>

#define R(i,j) R[(i) * matrices_dim + (j)]
#define A(i,j) A[(i) * matrices_dim + (j)]
#define B(i,j) B[(i) * matrices_dim + (j)]

// 
static void binary_reduce(matrix A, matrix B, matrix R, int matrices_dim)
{
    for (int i = 0; i < matrices_dim; i++)
        for (int j = 0; j < matrices_dim; j++)
            R(i,j) = fmin(A(i,j), B(i,j));
}

void sequential_reduce(matrix *matrices, int matrices_num, matrix result,
                       int matrices_dim)
{
    binary_reduce(matrices[0], matrices[1], result, matrices_dim);

    for (int i = 2; i < matrices_num; i++)
        binary_reduce(result, matrices[i], result, matrices_dim);
}

//
__inline__ __device__ int32_t warp_reduce_min(int32_t val)
{
    printf("%d ", val);

    for (int offset = warpSize / 2; offset > 0; offset /= 2)
        val = min(val, __shfl_down_sync(val, offset, warpSize));

    return val;
}

//
__inline__ __device__ int32_t block_reduce_min(int32_t val)
{
    static __shared__ int32_t shared[32];
    int laneid, warpid;

    laneid = threadIdx.x % warpSize;
    warpid = threadIdx.x / warpSize;

    val = warp_reduce_min(val);
    if (laneid == 0) shared[warpid] = val;
    __syncthreads();

    val = (threadIdx.x < blockDim.x / warpSize) ? shared[laneid] : INT32_MAX;
    if (warpid == 0) val = warp_reduce_min(val);

    return val;
}

// 
__global__ void cuda_reduce_kernel(int32_t *in, int32_t *out, int n)
{
    int minimum = INT32_MAX;

    for (int i = blockIdx.x * blockDim.x + threadIdx.x;
         i < n;
         i += blockDim.x * gridDim.x)
    {
        minimum = in[i];
    }

    minimum = block_reduce_min(minimum);

    if (threadIdx.x == 0)
        out[blockIdx.x] = minimum;
}

void cuda_reduce(matrix *matrices, int matrices_num, matrix result,
                 int matrices_dim)
{
    int32_t **temp_matrices;
    int32_t *dev_vector, *dev_result, *h_result;
    size_t in_size, out_size;
    int vector_rows, vector_cols, threads_per_block, blocks_per_grid;

    vector_rows = matrices_dim * matrices_dim;
    vector_cols = matrices_num;
    temp_matrices = (int32_t **) malloc(vector_rows * sizeof (int32_t *));

    while (vector_cols % 32 != 0)
        vector_cols++;

    threads_per_block = min(vector_cols, 1024);
    in_size = max(threads_per_block, vector_cols) * sizeof (int32_t);

    for (int i = 0; i < vector_rows; i++)
    {
        blocks_per_grid = (vector_cols / threads_per_block) + 1;
        out_size = blocks_per_grid * sizeof (int32_t);
        temp_matrices[i] = (int32_t *) malloc(in_size);

        for (int j = 0; j < matrices_num; j++)
            temp_matrices[i][j] = matrices[j][i];

        for (int j = matrices_num; j < vector_cols; j++)
            temp_matrices[i][j] = INT32_MAX;

        h_result = (int32_t *) malloc(out_size);
        hipMalloc(&dev_vector, in_size);
        hipMalloc(&dev_result, out_size);
        hipMemcpy(dev_vector, temp_matrices[i], in_size,
                   hipMemcpyHostToDevice);
        cuda_reduce_kernel<<<blocks_per_grid, threads_per_block>>>
            (dev_vector, dev_result, vector_cols);
        cuda_reduce_kernel<<<1, blocks_per_grid>>>
            (dev_result, dev_result, blocks_per_grid);
        hipMemcpy(h_result, dev_result, out_size, hipMemcpyDeviceToHost);
printf("%d \n", h_result[0]);
        hipFree(dev_vector);
        hipFree(dev_result);
    }

    for (int i = 0; i < vector_rows; i++)
        free(temp_matrices[i]);

    free(temp_matrices);
}
